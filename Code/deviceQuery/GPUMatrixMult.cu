#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <stdlib.h>

#include <sys/time.h>     //measuring performance data

_global__
void kernel_syr2k(int N, int M, double *C, double *A, double *B){
}



// Function to form upper triangular marix
void upper(int matrix[][N]){
    int i, j;
    for (i=0; i<N; i++)    {
        for (j=0; j<N; j++)    {
            if (i>j)   {
                matrix[i][j] = 0;
            }
            matrix[i][j] = matrix[i][j];
        }
    }
}



// This function multiplies mat1[][] and mat2[][],
// and stores the result in res[][]
void multiply(float mat1[][N], float mat2[][N], float res[][N])
{
    int i, j, k;
    for (i = 0; i < N; i++)
    {
        for (j = i; j < N; j++)
        {
            res[i][j] = 0.0;
            for (k = 0; k < N; k++)
                res[i][j] += mat1[i][k]*mat2[k][j];
        }
    }
}

void init_matricies(int N){
   
  double* mat1;
  double* mat2;
  double* ret;
  hipMallocManaged(&mat1, N*N * sizeof(float));		//cuda allocation of unified Memory  
  hipMallocManaged(&mat2, N*N * sizeof(float));
  hipMallocManaged(&ret, N * sizeof(float)); 
  
  upper(mat1); // sets the matrix to its upper bound
  upper(mat2); // sets the matrix to its upper bound

}


int main(int argc, char** argv)
{
    int N = 3;  // THIS IS WHERE YOU DEFINE THE SIZE OF THE MATRIX (it can only be square right now)
    //performance measurment 
    struct timeval t_start;
    struct timeval t_end;
    double etime;
    
    init_matricies(N);
    
    int i,j;
    multiply(mat1, mat2, res); // multiplies the 2 matricies 
    
    // print matrix 1
    printf("mat1 matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", mat1[i][j]);
        printf("\n");
    }
    // print matrix 2
    printf("mat2 matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", mat2[i][j]);
        printf("\n");
    }
    // print result matrix
    printf("Result matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", res[i][j]);
        printf("\n");
    }
    
    return 0;
}






int main(int argc, char** argv)
{
   
}
