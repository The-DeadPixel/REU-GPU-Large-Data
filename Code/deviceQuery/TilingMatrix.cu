/*
 *  file name: TilingMatrixV2.c
 *  NOTE: 
 *       squareMatrixMult is much more efficent than the regular multiplier
 *       currently compiling with: nvcc TilingMatrix.cu -o tileTest
 *       Device Standards for: GeForce GTX 1060 6GB
 *          total global mem size: 6078 MBytes (6373572608 bytes)
 *          total shared mem per block: 49.152 KBytes (49152 bytes)
 *       nvcc TilingMatrixV2.c -lcublas -o TilingMatrixTest
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <sys/time.h>     //measuring performance data

#define BLOCK_SIZE 32

/**********************************************************************
 * function name: matrixTriUpper
 * description: sets a matrix to an upper bound triangle matrix
 * parameters: 
 *            &a GPU device pointer to a m X n matrix (A)
 * Note:
 *    
 * return: none
 **********************************************************************/
__global__ void matrixTriUpper(float *a, int m, int n) {
    //setting matricies to their upper bound 
    for(int i = 0; i < m; ++i) {
        for(int j = 0; j < n; ++j) {
            if(i>j)
                a[i*n + j] = 0;
            a[i*n + j] = a[i*n + j];
        }
    }
}

/**********************************************************************
 * function name: cublasMatrixMult
 * description: dot product of two matricies using cublas function: cublasSgemm
 * parameters: 
 *            &a GPU device pointer to a m X m matrix (A)
 *            &b GPU device pointer to a n X k matrix (B)
 *            &c GPU device output purpose pointer to a m X k matrix (C) 
 *            to store the result
 * Note:
 *    grid and block should be configured as:
 *        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
 *        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
 *    further sppedup can be obtained by using shared memory to decrease global memory access times
 * return: none
 **********************************************************************/
// __global__ void cublasMatrixMult(float *a, float *b, float *c, int m, int n, int k) {
//     // Set status variables
//     cudaError_t cudaStat; // cudaMalloc status
//      cublasStatus_t stat; // CUBLAS functions statusx
//     cublasHandle_t handle; // CUBLAS context
//     
//     // matracies on the device
//     float *d_a; // d_a - a on the device
//     float *d_b; // d_b - b on the device
//     float *d_c; // d_c - c on the device
//     cudaStat = cudaMalloc((void**)&d_a,m*k*sizeof(*a)); // device memory alloc for a
//     cudaStat = cudaMalloc((void**)&d_b,k*n*sizeof(*b)); // device memory alloc for b
//     cudaStat = cudaMalloc((void**)&d_c,m*n*sizeof(*c)); // device memory alloc for c
//     //cudaGetErrorString((cudaError_t) cudaStat);
//     if(cudaStat != cudaSuccess)
//         printf("Cuda Error: %s\n", cudaGetErrorString(cudaStat));
//     
//     stat = cublasCreate(&handle); // initialize CUBLAS context
//     // copy matrices from the host to the device
//     stat = cublasSetMatrix(m,k,sizeof(*a),a,m,d_a,m); //a -> d_a
//     stat = cublasSetMatrix(k,n,sizeof(*b),b,k,d_b,k); //b -> d_b
//     stat = cublasSetMatrix(m,n,sizeof(*c),c,m,d_c,m); //c -> d_c
// //     if(stat != CUBLAS_STATUS_SUCCESS)
// //         printf("Cublas Error: %s\n", cublasGetErrorString(stat));
//     
//     float al =1.0f; // al=1
//     float bet =1.0f; // bet=1
//     // matrix-matrix multiplication: d_c=al*d_a*d_b+bet*d_c
//     // d_a -mxk matrix, d_b -kxn matrix, d_c -mxn matrix;
//     // al, bet -scalars
//     stat = cublasSgemm(handle,CUBLAS_OP_N,CUBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);
//     stat = cublasGetMatrix(m,n,sizeof(*c),d_c,m,c,m); // cp d_c -> c
// //     if(stat != CUBLAS_STATUS_SUCCESS)
// //         printf("Cublas Error: %s\n", cublasGetErrorString(stat));
//     
//     // free device memory
//     cudaFree(d_a);
//     cudaFree(d_b);
//     cudaFree(d_c);
//     cublasDestroy(handle); // destroy CUBLAS context
// }
/**************************************************************
 * function name: storeC
 * description: copies the final answers of tileC back to the corresponding indices of of Matrix C
 * 
 * parameters:
 *            &tilec          pointer to pre-allocated (tileLength X tileLength) matrix
 *            &matrixc        pointer to large (m X m)matrix B 
 *            int tileLength  predefined length of tile
 *            int i           caller outer loop value (helps define starting ROW index for tile)
 *            int j           caller inner loop value (helps define starting COLUMN for tile)
 * 
 ****************************************************************/
void storeC (float *tileC, float *matrixC, int tileLength, int i, int j, int numTiles){
    //pointer declarations
    for(int Ti = (tileLength*i); Ti < (tileLength*i)+tileLength; Ti++){
        for(int Tj = (tileLength*j); Tj < (tileLength*j) + tileLength; Tj++ ){
            matrixC[(Ti * tileLength) + Tj] = tileC[(Ti *numTiles *tileLength)+Tj];  
            printf("[%0.1f] ", matrixC[(Ti*tileLength) + Tj]);
        }
        printf("\n");
    }
    printf("\n");
    
    
}
/**************************************************************
 * function name: fillA
 * description: populates TileA with elements of matrix A that correspond to the to the correct starting indices and boundries of the tile.
 * 
 * parameters:
 *            &tileA          pointer to pre-allocated tileLength X tileLength matrix
 *            &matrixA        pointer to large matrix A 
 *            int tileLength  predefined length of tile
 *            int i           caller outer loop value (helps define starting ROW index for tile)
 *            int j           caller inner loop value (helps define starting COLUMN for tile)
 * 
 ****************************************************************/
void fillA(float *tileA, float *matrixA, int tileLength, int i, int j, int numTiles){
    for(int Ti = (tileLength*i); Ti < (tileLength*i)+tileLength; Ti++){
        for(int Tj = (tileLength*j); Tj < (tileLength*j) + tileLength; Tj++ ){
            tileA[(Ti * tileLength) + Tj] = matrixA[(Ti *numTiles *tileLength) + Tj]; 
            printf("[%0.1f] ", tileA[(Ti * tileLength) + Tj]);
        }
        printf("\n");
    }
    printf("\n");
    
    
}

/**************************************************************
 * function name: fillB
 * description: populates TileB with elements of matrix B that correspond to the to the correct starting indices and boundries of the
 * tile.
 * 
 * parameters:
 *            &tileB          pointer to pre-allocated (tileLength X tileLength) matrix
 *            &matrixB        pointer to large (m X m)matrix B 
 *            int tileLength  predefined length of tile
 *            int i           caller outer loop value (helps define starting COLUMN index for tile)
 *            int j           caller inner loop value (helps define starting ROW for tile)
 * 
 ****************************************************************/
void fillB(float *tileB, float *matrixB, int tileLength, int i, int j, int numTiles){
    //pointer declarations
    
    for(int Ti = (tileLength*j); Ti < (tileLength*j)+tileLength; Ti++){
        for(int Tj = (tileLength*i); Tj < (tileLength*i) + tileLength; Tj++ ){
            
            
            tileB[Ti * tileLength + Tj] = matrixB[Ti * numTiles* tileLength + Tj]; 
            printf("[%0.1f] ", tileB[Ti * tileLength + Tj]);
        }
        printf("\n");
    }
    printf("\n");
}

/**********************************************************************
 * function name: matrixCpy
 * description: Iterates through large (m X m) matricies A and B, continually creating smaller (tileLength * tileLength) matricies Ta and Tb that will be used by device to produce matrix C containing computed answers of MM of matrices A and B.  
 * parameters: 
 *            &a              GPU device pointer to a m X m matrix (A)
 *            &b              GPU device pointer to a m X m matrix (B)
 *            &c              GPU device output purpose pointer to a m X m matrix (C) 
 *            int tileLength  predefined max length of tile
 *            int m           # of tiles that divide the length of matrices A & B
 *        
 * return: none
 * TODO implement kernel calls of cuBlas and TMM, implement another function or code that tranfers results of C tile to matrix C. FIGURE OUT WHY fillA and fillB piss off the compiler
 **********************************************************************/
void matrixCpy(float *a, float *b, float *c, int tileLength, int m ) {
    // device and host TILE memory declaration
    hipError_t cudaStat;
    float *Ta,*Tb,*Tc, *d_a, *d_b, *d_c;
    int storeCHelper = 0;
    //    cublasHandle_t handle;
    for(int i = 0; i < m; i++)
    {
        
        //host and device tile allocation of C
        Tc = (float*) malloc(tileLength*tileLength* sizeof(float)); // host tile memory for c
        cudaStat = hipMalloc((void**)&d_c,tileLength*tileLength*sizeof(*c)); // device memory alloc for c
        if(cudaStat != hipSuccess)
            printf("Cuda malloc Error: %s\n", hipGetErrorString(cudaStat));
        //memcpy of tile C for host to device (POSSIBLE AREA FOR TIMING)
        cudaStat = hipMemcpy(d_c, Tc, tileLength*tileLength*sizeof(float), hipMemcpyHostToDevice);
        if(cudaStat != hipSuccess)
            printf("Cuda malloc Error: %s\n", hipGetErrorString(cudaStat));
        
        for(int j = 0; j < m; j++)
        {
            
            
            
            //Host memory alocation
            Ta = (float*) malloc(tileLength*tileLength* sizeof(float)*4); // host tile memory alloc for a
            Tb = (float*) malloc(tileLength*tileLength* sizeof(float)*4); // host tile memory alloc for b
            //Device memory allocation
            cudaStat = hipMalloc((void**)&d_a,tileLength*tileLength*sizeof(*a)); // device memory alloc for a
            if(cudaStat != hipSuccess)
                printf("Cuda A Malloc: %s\n", hipGetErrorString(cudaStat));
            cudaStat = hipMalloc((void**)&d_b,tileLength*tileLength*sizeof(*b)); // device memory alloc for b
            if(cudaStat != hipSuccess)
                printf("Cuda B Malloc: %s\n", hipGetErrorString(cudaStat));
            
            //Fill tileA & tileB with elements from matrix A (COMPILER IS SAYING THESE ARE UNDEFINED WTF) (POSSIBLE AREA FOR TIMING)
            printf("Tile A iteration: i=%d, j=%d\n", i,j);
            fillA(Ta, a, tileLength, i, j, m);
            printf("Tile B iteration: i=%d, j=%d\n", i,j);
            fillB(Tb, b, tileLength, i, j, m);
            //memcpy TileA and TileB froim host to device
            cudaStat = hipMemcpy(d_a, Ta, tileLength*tileLength*sizeof(float),hipMemcpyHostToDevice);
            if(cudaStat != hipSuccess)
                printf("Cuda memcpy: %s\n", hipGetErrorString(cudaStat));
            cudaStat = hipMemcpy(d_b, Tb, tileLength*tileLength*sizeof(float),hipMemcpyHostToDevice);
            if(cudaStat != hipSuccess)
                printf("Cuda memcpy Error: %s\n", hipGetErrorString(cudaStat));
            
            //Free device and host memory for next iteration
            printf("Free da\n");   
            cudaStat = hipFree(d_a);
            if(cudaStat != hipSuccess)
                printf("Cuda free Error: %s\n", hipGetErrorString(cudaStat));
            printf("Free db\n");   
            hipFree(d_b);
            if(cudaStat != hipSuccess)
                printf("Cuda free Error: %s\n", hipGetErrorString(cudaStat));
            printf("Free Ta\n");   
            //free(Ta);
            printf("Free Tb\n"); 
            //free(Tb);
            
        }
        
        //destroyHandle(handle);
        //memcpy c results back to host
        hipMemcpy(Tc,d_c, tileLength*tileLength*sizeof(float),hipMemcpyDeviceToHost);
        //copy tileC results back to matrix C 
        storeC(Tc,c, tileLength, i, storeCHelper, m);
        storeCHelper++;
        //Free device and host memory of C related arrays
        hipFree(d_c);
        free(Tc);
    }
}




/**********************************************************************
 * function name: cublasGetErrorString
 * description: gets the cublas string error codes for printing
 * parameters: 
 *            error a cublas error status enum
 * return: char pointer (string)
 * TODO: Fix the return type
 **********************************************************************/
const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "CUBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "CUBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "CUBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}


/**********************************************************************
 * function name: main
 * description: test and compare
 * parameters: 
 *            none
 * return: none
 **********************************************************************/
int main(int argc, char** argv) {
    int m=8;// a - mxk matrix
    int n=8;// b - kxn matrix
    int k=8;// c - mxn matrix
    // Set status variables
    
    // Allocate memory in host RAM
    float *a; // mxk matrix a on the host
    float *b; // kxn matrix b on the host
    float *c; // mxn matrix c on the host
    a = (float*) malloc(m*k* sizeof(float)); // host memory for a
    b = (float*) malloc(k*n* sizeof(float)); // host memory for b
    c = (float*) malloc(m*n* sizeof(float)); // host memory for c
    
    /* Assign Random Variables to the matrecies */
    //     srand(3333);
    int val = 1;
    // random initialize matrix A [mxk]
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] =val++;
        }
    }
    val = 1;
    // random initialize matrix B [kxn]
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = val++;
        }
    }
    
    //     // on host set the two matracies to triangles
    //     unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //     unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //     dim3 dimGrid(grid_cols, grid_rows);
    //     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    printf("Calculating...\n\n");
    // Launch kernel
    matrixCpy(a,b,c,4,2);
    //     matrixTriUpper<<<dimGrid, dimBlock>>>(a, m, n);
    //     matrixTriUpper<<<dimGrid, dimBlock>>>(b, n, k);
    //     cublasMatrixMult<<<dimGrid, dimBlock>>>(a,b,c,m,n,k);
    
    int i,j;
    // print matrix A
    printf("matA matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            //printf("[%d][%d]:%d, ", i, j, a[i*k + j]);
            printf(" %f ", a[i*k + j]);
        }
        printf("\n");
    }
    // print matrix B
    printf("\nmatB matrix: \n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < k; j++) {
            //printf("[%d][%d]:%d, ", i, j, b[i*k + j]);
            printf(" %f ", b[i*k + j]);
        }
        printf("\n");
    }
    // print result matrix
    printf("\nResult matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < k; j++) {
            //printf("[%d][%d]:%d, ", i, j, c[i*k + j]);
            printf(" %f ", c[i*k + j]);
        }
        printf("\n");
    }
    
    // free memory
    free(a);
    free(b);
    free(c);
    
    
    return 0;
}
